#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace std;


#define N 1024
#define BLOCK_SIZE 32

__global__ void matrixVectorMul(double* A, double* x, double* y) {
    __shared__ double s_x[BLOCK_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load vector x into shared memory
    s_x[tx] = x[tx];

    __syncthreads();

    // Compute dot product of row i and vector x
    double sum = 0.0;
    for (int j = 0; j < N; j++) {
        sum += A[i * N + j] * s_x[j % BLOCK_SIZE];
    }

    // Store result in vector y
    y[i] = sum;
}

int main() {
    double* A, * x, * y;
    double* d_A, * d_x, * d_y;

    // Allocate memory on host
    A = (double*)malloc(N * N * sizeof(double));
    x = (double*)malloc(N * sizeof(double));
    y = (double*)malloc(N * sizeof(double));

    // Initialize input data
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() / (double)RAND_MAX;
    }
    for (int i = 0; i < N; i++) {
        x[i] = rand() / (double)RAND_MAX;
        y[i] = 0.0;
    }

    printf("The matrix is:\n");
    printf("***************");
    for (int i = 0;i < N * N;i++) {
        if (i % N == 0) {

            printf("\n");
        }
        cout << A[i] << " ";
    }
    printf("\n\n");

    printf("The vector is:\n");
    printf("***************\n");
    for (int i = 0;i < N;i++) {
        cout << x[i] << " ";
    }
    printf("\n\n");

    // Allocate memory on device
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_x, N * sizeof(double));
    hipMalloc(&d_y, N * sizeof(double));

    // Copy input data from host to device
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
    matrixVectorMul << <dimGrid, dimBlock >> > (d_A, d_x, d_y);

    // Copy output data from device to host
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    // Print result
    printf("The vector is:\n");
    printf("***************\n");
    for (int i = 0; i < N; ++i) {
        cout<<y[i];
    }
    printf("\n");


    // Free memory on device
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    // Free memory on host
    free(A);
    free(x);
    free(y);

    return 0;
}