#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "hipblas.h"

using namespace std;


#define N 35
#define BLOCK_SIZE 1024

__global__ void matrixVectorMul(double* A, double* x, double* y) {
    __shared__ double s_x[BLOCK_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load vector x into shared memory
    s_x[tx] = x[tx];

    __syncthreads();

    // Compute dot product of row i and vector x
    double sum = 0.0;
    for (int j = 0; j < N; j++) {
        sum += A[i * N + j] * s_x[j % BLOCK_SIZE];
    }

    // Store result in vector y
    y[i] = sum;
}

int main() {
    double* A, * x, * y;
    double* d_A, * d_x, * d_y;

    // Allocate memory on host
    A = (double*)malloc(N * N * sizeof(double));
    x = (double*)malloc(N * sizeof(double));
    y = (double*)malloc(N * sizeof(double));

    // Initialize input data
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() / (double)RAND_MAX;
    }
    for (int i = 0; i < N; i++) {
        x[i] = rand() / (double)RAND_MAX;
        y[i] = 0.0;
    }

    printf("The matrix is:\n");
    printf("***************");
    for (int i = 0;i < N * N;i++) {
        if (i % N == 0) {

            printf("\n");
        }
        cout << A[i] << " ";
    }
    printf("\n\n");

    printf("The vector is:\n");
    printf("***************\n");
    for (int i = 0;i < N;i++) {
        cout << x[i] << " ";
    }
    printf("\n\n");

    // Allocate memory on device
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_x, N * sizeof(double));
    hipMalloc(&d_y, N * sizeof(double));

    // Copy input data from host to device
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    dim3 dimGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
    matrixVectorMul << <dimGrid, dimBlock >> > (d_A, d_x, d_y);

    // Copy output data from device to host
    hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    // Print result
    printf("The result vector is:\n");
    printf("***************\n");
    for (int i = 0; i < N; ++i) {
        cout<<y[i] << " ";
    }
    printf("\n");


    double  *d_blasResult, *blasResult;
    hipMalloc(&d_blasResult, N * sizeof(double));
    blasResult = (double*)malloc(N * sizeof(double));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const double scale = 1;

    hipblasDgemv(handle, HIPBLAS_OP_T, N, N, &scale, d_A, N, d_x,
                1, &scale, d_blasResult, 1);
    hipMemcpy(blasResult, d_blasResult, N * sizeof(double), hipMemcpyDeviceToHost);
    hipblasDestroy(handle);

    printf("The  blas result vector is:\n");
    printf("***************\n");
    for (int i = 0; i < N; ++i) {
        cout<<blasResult[i] << " ";
    }
    printf("\n");
    // Free memory on device
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    // Free memory on host
    free(A);
    free(x);
    free(y);

    return 0;
}