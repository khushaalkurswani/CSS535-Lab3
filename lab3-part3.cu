
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h> 
#include <math.h>
#include <string>
#include <vector>
#include "hipblas.h"

using namespace std;

#define UNROLL_CONST 4;

// kernel function where each thread performs matrix-vector multiplication 
//		for their corresponding 4 elements of the result vector
__global__ void multiplyMV(double *matrix, double *vector, double *result, int N) 
{
	int row = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (row < N && row + 3 < N) 
	{	
		for (int i = 0; i < N; i++) 
		{
			result[row] += matrix[row * N + i] * vector[i];
            result[row + 1] += matrix[(row + 1) * N + i] * vector[i];
            result[row + 2] += matrix[(row + 2) * N + i] * vector[i];
            result[row + 3] += matrix[(row + 3) * N + i] * vector[i];
            
		}
	}
    else if (row < N)
    {
        int leftOver = N - row;
        for (int i = 0; i < N; i++) 
		{
            for (int j = 0; j < leftOver; j++)
            {
                result[row + j] += matrix[(row + j) * N + i] * vector[i];
            }
        }
    }
}

// kernel function where each thread performs matrix-vector multiplication 
//		for their corresponding element of the result vector after the 
//      offset index
__global__ void multiplyMVLeftOver(double *matrix, double *vector, 
    double *result, int N, int offset)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (row < N) 
    {
        for (int i = 0; i < N; i++) 
		{
			result[row] += matrix[row * N + i] * vector[i];
		}
    }    
}

// Returns a random double between 0.01 and 10
double generateRandDouble()
{
    return 10 * (double)rand() / (double)RAND_MAX + 0.01;
}

// Populates given array with random doubles
// array's length must match the passed in size parameter
void fillRandom(double *arr, int N)
{
    for (int i = 0; i < N; i++)
    {
        arr[i] = generateRandDouble();
    }
}

// Populates given array with zeros (empty array)
// array's length must match the passed in size parameter
void fillDefault(double *arr, int N)
{
    for (int i = 0; i < N; i++)
    {
        arr[i] = 0;
    }
}

// Subtracts the result vector array from blasResult vector array
// Stores the calculated difference in the residual vector array
// result, blasResult, and residual arrays' lenghths must match the passed in
//      N parameter
void calcResidual(double *result, double *blasResult, double *residual, int N)
{
    for (int i = 0; i < N; i++) 
    {
        residual[i] = blasResult[i] - result[i];
    }
}

bool isResidualSmall(double *residual, int N)
{
    for (int i = 0; i < N; i++) 
    {
        if (residual[i] > 0.0001)
        {
            return false;
        }
    }
    
    return true;
}

void printConfig(int N, int numBlocks, int numThreads) 
{
    cout << "Execution Configuration:" << endl;
    cout << "N = " << N << ", "
        << "Number of Blocks =  " << numBlocks << ", "
        << "Number of Threads Per Block = " << numThreads << endl;
    cout << endl;
}

// Print's matrix array's elements
// If matrix array has too many elements, then truncated matrix is printed
// matrix array is 1D array not a 2D array but logically represents a 2d
//      array such that each row is stored in order
// matrix array's length is N * N where N is the passed in parameter 
//		representing number of rows (same as number of columns)
void printMatrix(double *matrix, int N)
{
    bool tooLarge = false;

    // check if matrix is too large
    if (N > 10)
    {
        N = 10;
        tooLarge = true;
    }

    // print matrix name
    cout << "Matrix" << endl;

    // print matrix elements
    for (int j = 0; j < N; j++)
    {
        for (int k = 0; k < N; k++)
        {
            cout << matrix[j * N + k] << " ";
        }

        // add ellipsis to represent truncation if matrix too large
        if (tooLarge)
        {
            cout << " ...";
        }

        cout << endl;
    }

    // add ellipsis to represent truncation if matrix too large
    if (tooLarge)
    {
        cout << " ..." << endl;
    }

    cout << endl;
}


// prints the elements in the given array vec
//		array vec's length much match N where N 
//		is the number of elements in array
// Turncates array if too large
void printVec(double* vec, int N, string name) 
{
	bool tooLarge = false;
	if (N > 100) 
	{
		tooLarge = true;
		N = 100; // truncate array to 100 elements
	}

	cout << name << " :" << endl;
	for (int i = 0; i < N; i++) 
	{
		cout <<  vec[i] << "  ";
	}
	
	if (tooLarge) 
	{
		cout << " ... "; // elipsis represents truncated
	}
	cout << endl << endl;
}

// sets up the execution configuration in the configs
// each element in configs is a vector where 
//      1st element is number of elements, 
//      2nd element is number of blocks, and 
//      3rd element is number of threads per block
void setUpConfigs(vector<vector<int>> &configs)
{
	// 1024 elements, 5 blocks, and 205 threads per block
    vector<int> config1 = {1025, 2, 256};
    configs.push_back(config1);
    
    // 4095 elements, 12 blocks, 342 threads per block
	//vector<int> config2 = {4095, 12, 342}; 
    //configs.push_back(config2);
    
    // 12 elements, 12 blocks, 1 threads per block
	vector<int> config3 = {15, 1, 4}; 
    configs.push_back(config3);
    
    //8190/13 =630
    //vector<int> config4 = {8190, 13, 630}; 
    //configs.push_back(config4);
    
    //11585/200=58
    //vector<int> config5 = {11585, 200, 58}; 
    //configs.push_back(config5);
}

int main(int argc, char *argv[]) 
{
    // Set up execution configurations
    vector<vector<int>> configs;
    setUpConfigs(configs);
    
    // host copies of matrix, vector, result
    double *matrix, *vector, *result, *blasResult, *residual; 

    // device copies of matrix, vector, result
    double *d_matrix, *d_vector, *d_result, *d_blasResult;
  
    for (int i = 0; i < configs.size(); i++)
    {
        // get execution configuration
        int N = configs[i][0];
        int numBlocks = configs[i][1];
        int numThreads = configs[i][2];
        
        // allocate memory on host
        int vectorSize = N * sizeof(double);
        int matrixSize = N * N * sizeof(double);

        matrix = (double *)malloc(matrixSize);
        fillRandom(matrix, N*N);
        
        vector = (double *)malloc(vectorSize);
        fillRandom(vector, N);

        result = (double *)malloc(vectorSize);
        blasResult = (double *)malloc(vectorSize);
        residual = (double *)malloc(vectorSize);

        // allocate memory on device
        hipMalloc((void **)&d_matrix, matrixSize);
        hipMalloc((void **)&d_vector, vectorSize);
        hipMalloc((void **)&d_result, vectorSize);
        hipMalloc((void **)&d_blasResult, vectorSize);

        // Copy inputs to device
        hipMemcpy(d_matrix, matrix, matrixSize, hipMemcpyHostToDevice);
        hipMemcpy(d_vector, vector, vectorSize, hipMemcpyHostToDevice);

        // lauch kernel function 
        multiplyMV<<<numBlocks, numThreads>>>(d_matrix, d_vector, d_result, N);
        
        //int completed = (N / 4) * 4;
        //int leftOver = N - completed;
        //multiplyMVLeftOver<<<1, leftOver>>>(d_matrix, d_vector, d_result, N, completed); 
        
        // Copy result back to host
        hipMemcpy(result, d_result, vectorSize, hipMemcpyDeviceToHost);

        // Calculate using cuBLAS
        // cuBLAS is column-major but matrix and vector are stored in row-major 
        //		so need to transpose matrix to ensure correct computation
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        const double scale = 1;
        hipblasDgemv(handle, HIPBLAS_OP_T, N, N, &scale, d_matrix, N, d_vector, 
                            1, &scale, d_blasResult, 1);
        hipMemcpy(blasResult, d_blasResult, vectorSize, hipMemcpyDeviceToHost);
        hipblasDestroy(handle);
        
        calcResidual(result, blasResult, residual, N);
        bool isSmallResidual = isResidualSmall(residual, N);

        // print all data
        printConfig(N, numBlocks, numThreads);
        printMatrix(matrix, N);
        printVec(vector, N, "Vector");
        printVec(result, N, "Kernel Result");
        printVec(blasResult, N, "cuBLAS Result");
        printVec(residual, N, "Residual");
        cout << "Is residual close to or equal to 0? ";
        if (isSmallResidual) 
        {
            cout << "Yes" << endl;
        }
        else 
        {
            cout << "No" << endl;
        }
            
        cout << endl << endl;

        // free memory on device
        hipFree(d_matrix);
        hipFree(d_vector);
        hipFree(d_result);
        hipFree(d_blasResult);
        
        // free memory on host
        free(matrix);
        free(vector);
        free(result);
        free(blasResult);
        free(residual);
    }

	return 0;
}
